
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <cassert>

using namespace nvcuda;
using namespace std;

static const int NUM_THREADS = 128;

//M,N must be multiples of BLOCK _M, BLOCK _N now
static const int M = 256, N = 160, K = 48;

//Cannot be changed now.
static const int BLOCK_M = 32;
static const int BLOCK_N = 32;

static const int WMMA_M = 16;
static const int WMMA_K = 16;
static const int WMMA_N = 16;

__global__ void wmma_matmul_kernel(__half *A, __half *B, __half*C) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, __half> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);

    /* 
    32x32 (128 threads), each warp takes 16x16
    w0 w1 
    w2 w3
    */
    int warp_id = threadIdx.x / 32;
    int m_warp = warp_id / 2; // Warp group row index
    int n_warp = warp_id % 2; // Warp group column index

    int A_block_offset = blockIdx.x * (BLOCK_M * K);
    int A_tile_offset = m_warp * (WMMA_M * K);
    __half *A_tile = A + A_block_offset + A_tile_offset;

    int B_block_offset = blockIdx.y * (BLOCK_N * K);
    int B_tile_offset = (n_warp * WMMA_N * K);
    __half *B_tile = B + B_block_offset + B_tile_offset;

    for (int tile_idx = 0; tile_idx < K; tile_idx += WMMA_K) {
        wmma::load_matrix_sync(a_frag, A_tile, K);
        wmma::load_matrix_sync(b_frag, B_tile, K);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        A_tile += WMMA_K;
        B_tile += WMMA_K;
    }

    int C_block_row_offset = blockIdx.x * BLOCK_M * N;
    int C_warp_row_offset = m_warp * WMMA_M * N;

    int C_block_col_offset = blockIdx.y * BLOCK_N;
    int c_warp_col_offset = n_warp * WMMA_N;
    
    __half *C_tile = C + C_block_row_offset + C_block_col_offset + C_warp_row_offset + c_warp_col_offset;
    wmma::store_matrix_sync(C_tile, c_frag, N, wmma::mem_row_major);
}


void cpu_matmul(const __half *A, const __half *B, float *C, int M, int N, int K) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += __half2float(A[i * K + k]) * __half2float(B[k * N + j]);
            }
            C[i * N + j] = sum;
        }
    }
}

void transpose_matrix(__half *src, __half *dest, int rows, int cols) {
    for (int row = 0; row < rows; row++) {
        for (int col = 0; col < cols; col++) {
            dest[col * rows + row] = src[row * cols + col];
        }
    }
}


int main() {
    __half h_A[M * K], h_B[K * N];
    float h_C[M * N], h_C_ref[M * N];

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            h_A[i * K + j] = rand() % 3;
        }
    }

    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            h_B[i * N + j] = rand() % 3;
        }
    }

    cout << "Matrix A:" << endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            cout << __half2float(h_A[i * K + j]) << " ";
        }
        cout << endl;
    }

    cout << "\nMatrix B:" << endl;
    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            cout << __half2float(h_B[i * N + j]) << " ";
        }
        cout << endl;
    }

    __half transposed_B[K * N];
    transpose_matrix(h_B, transposed_B, K, N);
    
    __half *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(__half));
    hipMalloc(&d_B, K * N * sizeof(__half));
    hipMalloc(&d_C, M * N * sizeof(__half));

    hipMemcpy(d_A, h_A, M * K * sizeof(__half), hipMemcpyHostToDevice);
    //cudaMemcpy(d_B, h_B, K * N * sizeof(__half), cudaMemcpyHostToDevice);
    hipMemcpy(d_B, transposed_B, K * N * sizeof(__half), hipMemcpyHostToDevice);

    dim3 threads(NUM_THREADS, 1, 1);
    //M,N
    int GRID_M = M / BLOCK_M;
    int GRID_N = N / BLOCK_N;
    dim3 blocks(GRID_M, GRID_N, 1);
    wmma_matmul_kernel<<<blocks, threads>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();

    __half h_C_half[M * N];
    hipMemcpy(h_C_half, d_C, M * N * sizeof(__half), hipMemcpyDeviceToHost);
    for (int i = 0; i < M * N; i++) h_C[i] = __half2float(h_C_half[i]);

    cpu_matmul(h_A, h_B, h_C_ref, M, N, K);

    cout << "\nMatrix C (GPU):" << endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cout << h_C[i * N + j] << " ";
        }
        cout << endl;
    }

    cout << "\nMatrix C (CPU Reference):" << endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cout << h_C_ref[i * N + j] << " ";
        }
        cout << endl;
    }

    for (int i = 0; i < M * N; i++) {
        //if( abs(h_C[i] - h_C_ref[i]) < 1e-2 ){
        //    cout << "our: " << h_C[i] << " ref: " << h_C_ref[i] << " " << i << "\n";
        //    assert(1);
        //}
        //cout << "our: " << h_C[i] << " ref: " << h_C_ref[i] << " " << i << "\n";
        assert(abs(h_C[i] - h_C_ref[i]) < 1e-2);
    }

    cout << "\nResults match!" << endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
