
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <cassert>

using namespace nvcuda;
using namespace std;

static const int NUM_THREADS = 256;

//M,N must be multiples of BLOCK _M, BLOCK _N now
static const int M = 512, N = 768, K = 512;

static const int WMMA_M = 16;
static const int WMMA_K = 16;
static const int WMMA_N = 16;

static const int WARPS_M = 4; // tunable
static const int M_PER_WARPS = WMMA_M * 2; // 32
static const int BLOCK_M = M_PER_WARPS * WARPS_M;

static const int WARPS_N = 2; // tunable
static const int N_PER_WARPS = WMMA_N * 4; // 64
static const int BLOCK_N = N_PER_WARPS * WARPS_N;


__global__ void wmma_matmul_kernel(__half *A, __half *B, __half*C) {
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, __half> c_frag;

    int warp_id = threadIdx.x / 32;
    int m_warp = warp_id / 4; // Warp group row index
    int n_warp = warp_id % 4; // Warp group column index

    int A_block_offset = blockIdx.x * (BLOCK_M * K);
    int A_warp_offset = m_warp * (WMMA_M * K);

    int B_block_offset = blockIdx.y * (BLOCK_N * K);
    int B_warp_offset = (n_warp * WMMA_N * K);

    for( int tile_m_idx = 0; tile_m_idx < WARPS_M; tile_m_idx++ ) {
        int A_tile_offset = tile_m_idx * (M_PER_WARPS * K);

        for( int tile_n_idx = 0; tile_n_idx < WARPS_N; tile_n_idx++ ) {
            __half *A_tile = A + A_block_offset + A_warp_offset + A_tile_offset;

            int B_tile_offset = tile_n_idx * (N_PER_WARPS * K);
            __half *B_tile = B + B_block_offset + B_warp_offset + B_tile_offset;
            wmma::fill_fragment(c_frag, 0.0f);

            for (int tile_idx = 0; tile_idx < K; tile_idx += WMMA_K) {
                wmma::load_matrix_sync(a_frag, A_tile, K);
                wmma::load_matrix_sync(b_frag, B_tile, K);
                wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
                A_tile += WMMA_K;
                B_tile += WMMA_K;
            }

            int C_block_row_offset = blockIdx.x * BLOCK_M * N;
            int C_warp_row_offset = m_warp * WMMA_M * N;
            int C_tile_row_offset = tile_m_idx * M_PER_WARPS * N;

            int C_block_col_offset = blockIdx.y * BLOCK_N;
            int c_warp_col_offset = n_warp * WMMA_N;
            int C_tile_col_offset = tile_n_idx * N_PER_WARPS;

            __half *C_tile = C + C_block_row_offset + C_block_col_offset +
                                C_warp_row_offset + c_warp_col_offset +
                                C_tile_row_offset + C_tile_col_offset;
            wmma::store_matrix_sync(C_tile, c_frag, N, wmma::mem_row_major);
        }
    }
}

void print_matrix(const __half *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            cout << __half2float(matrix[i * cols + j]) << " ";
        }
        cout << endl;
    }
}

void print_matrix(const float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            cout << matrix[i * cols + j] << " ";
        }
        cout << endl;
    }
}

void cpu_matmul(const __half *A, const __half *B, float *C, int M, int N, int K) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += __half2float(A[i * K + k]) * __half2float(B[k * N + j]);
            }
            C[i * N + j] = sum;
        }
    }
}

void transpose_matrix(__half *src, __half *dest, int rows, int cols) {
    for (int row = 0; row < rows; row++) {
        for (int col = 0; col < cols; col++) {
            dest[col * rows + row] = src[row * cols + col];
        }
    }
}

void fill_random_matrix(__half *matrix, int rows, int cols, int max_value = 3) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            matrix[i * cols + j] = rand() % max_value;
        }
    }
}


int main() {
    __half *h_A = new __half[M * K];
    __half *h_B = new __half[K * N];
    float *h_C = new float[M * N];
    float *h_C_ref = new float[M * N];

    fill_random_matrix(h_A, M, K);
    fill_random_matrix(h_B, K, N);

    __half *transposed_B = new __half[K * N];
    transpose_matrix(h_B, transposed_B, K, N);
    
    __half *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(__half));
    hipMalloc(&d_B, K * N * sizeof(__half));
    hipMalloc(&d_C, M * N * sizeof(__half));

    hipMemcpy(d_A, h_A, M * K * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, transposed_B, K * N * sizeof(__half), hipMemcpyHostToDevice);

    dim3 threads(NUM_THREADS, 1, 1);
    int GRID_M = M / BLOCK_M;
    int GRID_N = N / BLOCK_N;
    dim3 blocks(GRID_M, GRID_N, 1);
    cout << "GRID M : " << GRID_M << " GRID_N : " << GRID_N <<  endl;
    int CNT = 1;
    for(int i = 0; i < CNT; i++){
        wmma_matmul_kernel<<<blocks, threads>>>(d_A, d_B, d_C);
    }
    hipDeviceSynchronize();

    __half *h_C_half = new __half[M * N];
    hipMemcpy(h_C_half, d_C, M * N * sizeof(__half), hipMemcpyDeviceToHost);
    for (int i = 0; i < M * N; i++) h_C[i] = __half2float(h_C_half[i]);

    bool cpu_test = true;
    if (cpu_test) {
        cout << "Matrix A:" << endl;
        print_matrix(h_A, M, K);

        cout << "\nMatrix B:" << endl;
        print_matrix(h_B, K, N);

        cpu_matmul(h_A, h_B, h_C_ref, M, N, K);

        cout << "\nMatrix C (GPU):" << endl;
        print_matrix(h_C, M, N);

        cout << "\nMatrix C (CPU Reference):" << endl;
        print_matrix(h_C_ref, M, N);

        for (int i = 0; i < M * N; i++) {
            assert(abs(h_C[i] - h_C_ref[i]) < 1e-2);
        }
        cout << "\nResults match!" << endl;
    }

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_ref;
    delete[] h_C_half;
    delete[] transposed_B;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
